#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdexcept>

#include <thrust/functional.h> // for binary_function
#include <thrust/host_vector.h> // for binary_function
#include <thrust/device_vector.h> // for binary_function
#include <functional>
struct BinaryOp: public thrust::binary_function<float,float,float> {
  sum(std::function<float(float,float) f) : m_f(f) { }
  __device__ float operator()(float x,float y) const {
    return f(x,y);
  }
};
float sum(float x,float y) { return x+y; }
void test ( struct sum *op) {
  const int N = 1<<10;

  thrust::host_vector<float> x_th_host_v;
  thrust::host_vector<float> y_th_host_v;
  for (int i = 0; i < N; i++) {
    x_th_host_v.push_back(i+0.0f);
    y_th_host_v.push_back(i+0.5f);
  }
  thrust::device_vector<float> x_th_dev_v(x_th_host_v);
  thrust::device_vector<float> y_th_dev_v(y_th_host_v);

  thrust::transform(x_th_dev_v.begin(),x_th_dev_v.end(),y_th_dev_v.begin(),y_th_dev_v.begin(),*op);

  y_th_host_v = y_th_dev_v;

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y_th_host_v[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;
}
void logLL() {
  test(new BinaryOp(sum));
}
void RPF() {
}
void MVLL() {
}
