
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ int const_symbol;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/***************/
/* TEST KERNEL */
/***************/
__global__ void kernel() {

    printf("Address of symbol from device = %p\n", &const_symbol);

}

/********/
/* MAIN */
/********/
int main() 
{
    const int N = 16;
    int *pointer = NULL;

    gpuErrchk(hipGetSymbolAddress((void**)&pointer, HIP_SYMBOL(const_symbol)));

    kernel<<<1,1>>>();

    printf("Address of symbol from host = %p\n", pointer);

    return 0;
}

