
#include <hip/hip_runtime.h>
#include <iostream>
typedef float (*fun)(float,float);
__host__ __device__ float sum(float x,float y) { return x+y; }
//__device__ fun m = sum;
__device__ float (*m)(float, float) = sum;
int main() {
  void *addr;
  hipError_t ret = hipGetSymbolAddress(&addr,HIP_SYMBOL(m));
  std::cout<<addr<<" "<<hipGetErrorString(ret)<<std::endl;

  return 0;
}
