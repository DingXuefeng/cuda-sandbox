#include <iostream>
#include <string>
#include <stdexcept>

#include <thrust/functional.h> // for binary_function
#include <thrust/host_vector.h> // for binary_function
#include <thrust/device_vector.h> // for binary_function

typedef float (*nvstdfunction)(float x,float y);

#define MEMCPY_TO_SYMBOL(target, source, count, offset, direction) \
  do { hipError_t ret = hipMemcpyToSymbol(HIP_SYMBOL(target), source, count, offset, direction); \
      if(ret!=hipSuccess) throw runtime_error(string(hipGetErrorString(ret))); } while(0)

#define TEST(OP_PTR)  \
  do { void *address; \
  hipError_t ret = hipGetSymbolAddress(&address, HIP_SYMBOL(OP_PTR ## _ptr)); \
  if(ret!=hipSuccess) { std::cout<<"on line "<<__LINE__<<std::endl; throw std::runtime_error(std::string(hipGetErrorString(ret))); } \
  test(new BinaryOp(address)); } while (0);

#include "fun.icc"

#define REGISTER(fun) \
__device__ nvstdfunction fun ## _ptr = fun;
REGISTER(logLLdev)
REGISTER(RPFdev)
REGISTER(MVLLdev20)
REGISTER(MVLLdev50)
REGISTER(MVLLdev70)

struct BinaryOp: public thrust::binary_function<float,float,float> {

  BinaryOp(void *address) { 
    hipError_t ret = hipMemcpy(&m_f, address, sizeof(void*),hipMemcpyDeviceToHost);
    if(ret!=hipSuccess) throw std::runtime_error(std::string(hipGetErrorString(ret)));
  }

  __device__ float operator()(float x,float y) const {
    return (*(reinterpret_cast<nvstdfunction>(m_f)))(x,y);
  }
  void *m_f;

};

void test ( struct BinaryOp*op) {
  const int N = 1<<10;

  thrust::host_vector<float> x_th_host_v;
  thrust::host_vector<float> y_th_host_v;
  for (int i = 0; i < N; i++) {
    x_th_host_v.push_back(1.0f);
    y_th_host_v.push_back(2.0f);
  }
  thrust::device_vector<float> x_th_dev_v(x_th_host_v);
  thrust::device_vector<float> y_th_dev_v(y_th_host_v);

  thrust::transform(x_th_dev_v.begin(),x_th_dev_v.end(),y_th_dev_v.begin(),y_th_dev_v.begin(),*op);

  y_th_host_v = y_th_dev_v;

//  std::cout<<y_th_host_v[0]<<std::endl;
}
void logLL() {
  TEST(logLLdev)
}
void RPF() {
  TEST(RPFdev)
}
void MVLL() {
  TEST(MVLLdev20)
  TEST(MVLLdev50)
  TEST(MVLLdev70)
}
