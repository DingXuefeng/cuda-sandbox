#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <iostream>
struct functor{
  functor(float (*g)(const float&)) : _g{g} {}

  __host__ __device__ float operator()(const float& x) const {
    return _g(x);
  }
  private:
  float (*_g)(const float&);
};

__host__ __device__ float g(const float& x){return 3*x;}

__device__ float (*d_g)(const float&) = g;

int main(void){
  float (*h_g)(const float&) = NULL;
  hipMemcpyFromSymbol(&h_g, HIP_SYMBOL(d_g), sizeof(void *));
  thrust::device_vector<float> X(4,1);
  thrust::transform(X.begin(), X.end(), X.begin(), functor(h_g));
  thrust::copy_n(X.begin(), X.size(), std::ostream_iterator<float>(std::cout, ","));
  std::cout << std::endl;
}
